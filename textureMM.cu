#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define CUDA_CHECK(X)                                                     \
    {                                                                     \
        hipError_t _m_cudaStat = X;                                      \
        if (hipSuccess != _m_cudaStat) {                                 \
            fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
                    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#define SIZE 8192 * 12 * 12
#define THREADSIZE 1024
#define BLOCKSIZE ((SIZE - 1) / THREADSIZE + 1)
#define RADIX 10
#define MAXSM 12
#define FILE_TO_OPEN "OURLASTCODE_shared_measures.csv"

__global__ void copyKernel(int *inArray, int *semiSortArray, int arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < arrayLength) {
        inArray[index] = semiSortArray[index];
    }
}
__global__ void reduceMaxMin(int *g_idata, int *g_maxdata, int *g_mindata) {
    __shared__ int smaxdata[(SIZE / BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(SIZE / BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    smaxdata[tid] = g_idata[i];
    smindata[tid] = g_idata[i];
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem

    if (tid == 0) {
        g_maxdata[blockIdx.x] = smaxdata[0];
        g_mindata[blockIdx.x] = smindata[0];
    }
}

__global__ void reduceMaxMin_Service(int *g_maxdata, int *g_mindata, int *max, int *min) {
    __shared__ int smaxdata[(THREADSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(THREADSIZE)];
    int tid = threadIdx.x;
    smaxdata[tid] = g_maxdata[tid];
    smindata[tid] = g_mindata[tid];
    for (unsigned int s = 1; s < BLOCKSIZE / THREADSIZE; s++) {
        int index = THREADSIZE * s + tid;
        if (smaxdata[tid] < g_maxdata[index])
            smaxdata[tid] = g_maxdata[index];
        if (smindata[tid] > g_mindata[index])
            smindata[tid] = g_mindata[index];
    }
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem
    if (tid == 0) {
        *max = smaxdata[0];
        *min = smindata[0];
    }
}

__global__ void resetBucket(int *bucket) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    bucket[index] = 0;
}

__global__ void histogramKernel(int *inArray, int *outArray, int *radixArray, int arrayLength, int significantDigit, int minElement) {
    __shared__ int inArrayShared[THREADSIZE];
    __shared__ int outArrayShared[RADIX];
    __shared__ int radixArrayShared[THREADSIZE];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread = threadIdx.x;
    int blockIndex = blockIdx.x * RADIX;
    if (index > arrayLength) {
        return;
    }
    int radix;
    int arrayElement;
    int i;

    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArrayShared[i] = 0;
        }
    }

    if (index < arrayLength) {
        inArrayShared[thread] = inArray[index];
    }

    __syncthreads();

    if (index < arrayLength) {
        arrayElement = inArrayShared[thread] - minElement;
        radix = ((arrayElement / significantDigit) % 10);
        radixArrayShared[thread] = radix;
        atomicAdd(&outArrayShared[radix], 1);
    }

    if (index < arrayLength) {
        radixArray[index] = radixArrayShared[thread];
    }
    __syncthreads();
    // forse possimao fare il casino che diventa supermegaultravelocissimo !!!!!!
    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[blockIndex + i] += outArrayShared[i];
        }
    }
}

__global__ void combineBucket(int *blockBucketArray, int *bucketArray, int block_size) {
    __shared__ int bucketArrayShared[RADIX];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index] = 0;

    for (i = index; i < RADIX * block_size; i = i + RADIX) {
        atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 1; i < RADIX; i++) {
            bucketArrayShared[i] += bucketArrayShared[i - 1];
        }
    }

    __syncthreads();
    bucketArray[index] = bucketArrayShared[index];
}

__global__ void indexArrayKernel(int *radixArray, int *bucketArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    int radix;
    int pocket;

    if (index < RADIX) {
        for (i = 0; i < arrayLength; i++) {
            radix = radixArray[arrayLength - i - 1];
            if (radix == index) {
                pocket = --bucketArray[radix];
                indexArray[arrayLength - i - 1] = pocket;
            }
        }
    }
}

__global__ void semiSortKernel(int *inArray, int *outArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;
    printf("\nalfredo\n");
    if (index < arrayLength) {
        arrayElement = inArray[index];
        arrayIndex = indexArray[index];
        outArray[arrayIndex] = arrayElement;
    }
}

void printArray(int *array, int size) {
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}

int findLargestNum(int *array, int size) {
    int i;
    int largestNum = -1;
    for (i = 0; i < size; i++) {
        if (array[i] > largestNum)
            largestNum = array[i];
    }
    return largestNum;
}
void make_csv(float gflops, float time, float N) {
    FILE *fp;
    if (access(FILE_TO_OPEN, F_OK) == 0) {
        fp = fopen(FILE_TO_OPEN, "a");

    } else {
        fp = fopen(FILE_TO_OPEN, "w");
        fprintf(fp, "N, gflops, time_sec\n");
    }
    fprintf(fp, "%f, %f, %.5f\n", N, gflops, time);
    fclose(fp);
}

void radixSort(int *array, int size) {
    int significantDigit = 1;
    hipEvent_t start, stop;
    int threadCount;
    int blockCount;

    int min, max;

    hipStream_t stream[MAXSM];

    for (int i = 0; i <= MAXSM; i++)
        hipStreamCreate(&stream[i]);
    threadCount = THREADSIZE;
    blockCount = BLOCKSIZE;
    int max_digit;
    // da calcolare bene
    int *outputArray;
    int *inputArray;
    int *radixArray;
    int *bucketArray;
    int *indexArray;
    int *semiSortArray;
    int *blockBucketArray;
    int *g_maxdata;
    int *g_mindata;
    int *largestNum;
    int *smallestNum;
    CUDA_CHECK(hipMalloc((void **)&inputArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&indexArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&g_maxdata, sizeof(int) * BLOCKSIZE));
    CUDA_CHECK(hipMalloc((void **)&g_mindata, sizeof(int) * BLOCKSIZE));

    CUDA_CHECK(hipMalloc((void **)&radixArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&outputArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&semiSortArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&bucketArray, sizeof(int) * RADIX));
    CUDA_CHECK(hipMalloc((void **)&blockBucketArray, sizeof(int) * RADIX * BLOCKSIZE));

    hipMemcpy(inputArray, array, sizeof(int) * size, hipMemcpyHostToDevice);

    hipMalloc((void **)&largestNum, sizeof(int));
    hipMalloc((void **)&smallestNum, sizeof(int));

    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    reduceMaxMin<<<blockCount, threadCount>>>(inputArray, g_maxdata, g_mindata);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    reduceMaxMin_Service<<<1, THREADSIZE>>>(g_maxdata, g_mindata, largestNum, smallestNum);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }

    hipMemcpy(&max, largestNum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, smallestNum, sizeof(int), hipMemcpyDeviceToHost);
    int new_size_first = size / MAXSM + size % MAXSM;
    int new_size_second = size / MAXSM;
    int my_size, offset = 0;
    int new_block_size;

    int *myradix = (int *)malloc(size * sizeof(int));
    int mycsoa = 1;
    int bucket[RADIX];
    max_digit = max - min;
    for (int j = 1; j <= MAXSM; j++) {
        if (j == 1) {
            hipMemcpyAsync(inputArray, array, new_size_first * sizeof(int), hipMemcpyHostToDevice, stream[j]);
            my_size = new_size_first;
            offset = 0;
        } else {
            hipMemcpyAsync(inputArray + new_size_second * (j - 1) + size % MAXSM, array + new_size_second * (j - 1) + size % MAXSM, new_size_second * sizeof(int), hipMemcpyHostToDevice, stream[j]);
            my_size = new_size_second;
            offset = new_size_second * (j - 1) + size % MAXSM;
        }
    }

    while (true) {
        for (int k = 0; k < RADIX; k++)
            bucket[k] = 0;
        printf(" ordino le %d\n", significantDigit);
        resetBucket<<<BLOCKSIZE, RADIX>>>(blockBucketArray);
        resetBucket<<<BLOCKSIZE, THREADSIZE>>>(semiSortArray);
        hipDeviceSynchronize();
        for (int j = 1; j <= MAXSM; j++) {
            if (j == 1) {
                my_size = new_size_first;
                offset = 0;
            } else {
                my_size = new_size_second;
                offset = new_size_second * (j - 1) + size % MAXSM;
            }

            new_block_size = (my_size - 1) / THREADSIZE + 1;
            hipMemcpyAsync(bucketArray, bucket, sizeof(int) * RADIX, hipMemcpyHostToDevice, stream[j]);
            histogramKernel<<<new_block_size, THREADSIZE, 0, stream[j]>>>(inputArray + offset, blockBucketArray, radixArray + offset, my_size, significantDigit, min);

            mycudaerror = hipGetLastError();
            if (mycudaerror != hipSuccess) {
                fprintf(stderr, "eheheh%s\n", hipGetErrorString(mycudaerror));
                exit(1);
            }

            // calcolo la frequenza per ogni cifra, sommando quelle di tutti i block.
            // fondamentalmente sommo all'array delle frequenze il precedente, come facevamo nel vecchio algortimo. A[i-1] = A[i]
            combineBucket<<<1, RADIX, 0, stream[j]>>>(blockBucketArray, bucketArray, new_block_size);
            hipDeviceSynchronize();
            mycudaerror = hipGetLastError();
            if (mycudaerror != hipSuccess) {
                fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
                exit(1);
            }
        }
        /* int *tuttublock = (int *)malloc(RADIX * BLOCKSIZE * sizeof(int));
         hipMemcpy(tuttublock, bucketArray, sizeof(int) * RADIX, hipMemcpyDeviceToHost);
         printf("\nblockbucketararay: ");
         for (int k = 0; k < RADIX; k++)
             printf(" %d ", tuttublock[k]);
         int arraybucket[10];
         for (int k = 0; k < 10; k++) {
             arraybucket[k] = 0;
         }
         printf("\nmiocoso:\t ");
         for (int k = 0; k < size; k++) {
             arraybucket[(array[k] / mycsoa) % 10]++;
         }
         mycsoa *= 10;
         for (int k = 1; k < 10; k++) {
             arraybucket[k] += arraybucket[k - 1];
         }
         for (int k = 0; k < 10; k++) {
             printf(" %d ", arraybucket[k]);
         }*/

        // reduce bucketArray
        // salva gli indici in cui memorizzare gli elementi ordinati --> fa la magia :D
        indexArrayKernel<<<BLOCKSIZE, THREADSIZE>>>(radixArray, bucketArray, indexArray, size, significantDigit);

        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        for (int j = 1; j <= MAXSM; j++) {
            if (j == 1) {
                my_size = new_size_first;
                offset = 0;
            } else {
                my_size = new_size_second;
                offset = new_size_second * (j - 1) + size % MAXSM;
            }
            printf("aahfleabeto");
            new_block_size = (my_size - 1) / THREADSIZE + 1;
            // salva gli elementi nella corretta posizione ordinati.
            semiSortKernel<<<new_block_size, THREADSIZE, 0, stream[j]>>>(inputArray + offset, semiSortArray, indexArray + offset, my_size, significantDigit);
        }

        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // aggiorno inputArray con il semisortedarray
        copyKernel<<<BLOCKSIZE, THREADSIZE>>>(inputArray, semiSortArray, size);

        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

        significantDigit *= RADIX;
        break;
    }
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);
    printf("ESCODALDOPPIOFOR");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float transferTime;
    hipEventElapsedTime(&transferTime, start, stop);
    printf("CUDA Time = %.5f ms dim=%d\n", transferTime, size);
    make_csv(0, transferTime, size);
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);
}

int main() {
    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int *array;
    hipHostMalloc((void **)&array, size * sizeof(int));
    int i;
    int max_digit = 9999;
    srand(time(NULL));

    for (i = 0; i < size; i++) {
        if (i % 2)
            array[i] = -(rand() % max_digit);
        else
            array[i] = (rand() % max_digit);
    }

    // printf("\nUnsorted List: ");
    // printArray(array, size);

    radixSort(array, size);
    for (int i = 1; i < size; i++)
        if (array[i - 1] > array[i]) {
            printf("SE SCASSATT O PUNTATOR");
            break;
        }

    // printf("\nSorted List:");
    // printArray(array, size);

    printf("\n");

    return 0;
}