#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define CUDA_CHECK(X)                                                     \
    {                                                                     \
        hipError_t _m_cudaStat = X;                                      \
        if (hipSuccess != _m_cudaStat) {                                 \
            fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
                    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#define SIZE 8192
#define THREADSIZE 64
#define BLOCKSIZE ((SIZE - 1) / THREADSIZE + 1)
#define RADIX 10
#define FILE_TO_OPEN "test2_texture_measures.csv"

texture<int, 1> texture_inputArray;

texture<int, 1> texture_radixArray;  // donotremove

__device__ float fetchInputArrayElement(int value) {
    return tex1Dfetch(texture_inputArray, value);
}
__device__ float fetch_radixArrayElement(int value) {
    return tex1Dfetch(texture_radixArray, value);
}

__global__ void copyKernel(int *inArray, int *semiSortArray, int arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < arrayLength) {
        inArray[index] = semiSortArray[index];
    }
}
__global__ void reduceMaxMin(int *g_maxdata, int *g_mindata) {
    __shared__ int smaxdata[(SIZE / BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(SIZE / BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    smaxdata[tid] = fetchInputArrayElement(i);
    smindata[tid] = fetchInputArrayElement(i);
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem

    if (tid == 0) {
        g_maxdata[blockIdx.x] = smaxdata[0];
        g_mindata[blockIdx.x] = smindata[0];
    }
}

__global__ void reduceMaxMin_Service(int *g_maxdata, int *g_mindata, int *max, int *min) {
    __shared__ int smaxdata[(THREADSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(THREADSIZE)];
    int tid = threadIdx.x;
    smaxdata[tid] = g_maxdata[tid];
    smindata[tid] = g_mindata[tid];
    for (unsigned int s = 1; s < BLOCKSIZE / THREADSIZE; s++) {
        int index = THREADSIZE * s + tid;
        if (smaxdata[tid] < g_maxdata[index])
            smaxdata[tid] = g_maxdata[index];
        if (smindata[tid] > g_mindata[index])
            smindata[tid] = g_mindata[index];
    }
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem
    if (tid == 0) {
        *max = smaxdata[0];
        *min = smindata[0];
    }
}

__global__ void histogramKernel(int *outArray, int *radixArray, int arrayLength, int significantDigit, int minElement) {
    __shared__ int inArrayShared[THREADSIZE];
    __shared__ int outArrayShared[RADIX];
    __shared__ int radixArrayShared[THREADSIZE];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread = threadIdx.x;
    int blockIndex = blockIdx.x * RADIX;

    int radix;
    int arrayElement;
    int i;

    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArrayShared[i] = 0;
        }
    }

    if (index < arrayLength) {
        inArrayShared[thread] = fetchInputArrayElement(index);
    }

    __syncthreads();

    if (index < arrayLength) {
        arrayElement = inArrayShared[thread] - minElement;
        radix = ((arrayElement / significantDigit) % 10);
        radixArrayShared[thread] = radix;
        atomicAdd(&outArrayShared[radix], 1);
    }

    if (index < arrayLength) {
        radixArray[index] = radixArrayShared[thread];
    }
    __syncthreads();
    // forse possimao fare il casino che diventa supermegaultravelocissimo !!!!!!
    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[blockIndex + i] = outArrayShared[i];
        }
    }
}

__global__ void combineBucket(int *blockBucketArray, int *bucketArray) {
    __shared__ int bucketArrayShared[RADIX];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index] = 0;

    for (i = index; i < RADIX * BLOCKSIZE; i = i + RADIX) {
        atomicAdd(&bucketArrayShared[index], bucketArray[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 1; i < RADIX; i++)
            bucketArrayShared[i] += bucketArrayShared[i - 1];
    }
    __syncthreads();
    bucketArray[index] = bucketArrayShared[index];
}

__global__ void indexArrayKernel(int *bucketArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    int radix;
    int pocket;

    if (index < RADIX) {
        for (i = 0; i < arrayLength; i++) {
            radix = fetch_radixArrayElement(arrayLength - i - 1);
            if (radix == index) {
                pocket = --bucketArray[radix];
                indexArray[arrayLength - i - 1] = pocket;
            }
        }
    }
}

__global__ void semiSortKernel(int *outArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;

    if (index < arrayLength) {
        arrayElement = fetchInputArrayElement(index);
        arrayIndex = indexArray[index];
        outArray[arrayIndex] = arrayElement;
    }
}

void printArray(int *array, int size) {
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}

int findLargestNum(int *array, int size) {
    int i;
    int largestNum = -1;
    for (i = 0; i < size; i++) {
        if (array[i] > largestNum)
            largestNum = array[i];
    }
    return largestNum;
}
void make_csv(float gflops, float time, float N) {
    FILE *fp;
    if (access(FILE_TO_OPEN, F_OK) == 0) {
        fp = fopen(FILE_TO_OPEN, "a");

    } else {
        fp = fopen(FILE_TO_OPEN, "w");
        fprintf(fp, "N, gflops, time_sec\n");
    }
    fprintf(fp, "%f, %f, %.5f\n", N, gflops, time);
    fclose(fp);
}

void radixSort(int *array, int size) {
    int significantDigit = 1;
    hipEvent_t start, stop;
    int threadCount;
    int blockCount;

    int min, max;

    threadCount = THREADSIZE;
    blockCount = BLOCKSIZE;

    int *outputArray;
    int *inputArray;
    int *radixArray;
    int *bucketArray;
    int *indexArray;
    int *semiSortArray;
    int *blockBucketArray;
    int *g_maxdata;
    int *g_mindata;
    int *largestNum;
    int *smallestNum;

    CUDA_CHECK(hipMalloc((void **)&inputArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&indexArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&g_maxdata, sizeof(int) * BLOCKSIZE));
    CUDA_CHECK(hipMalloc((void **)&g_mindata, sizeof(int) * BLOCKSIZE));

    CUDA_CHECK(hipMalloc((void **)&radixArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&outputArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&semiSortArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&bucketArray, sizeof(int) * RADIX));
    CUDA_CHECK(hipMalloc((void **)&blockBucketArray, sizeof(int) * RADIX * BLOCKSIZE));

    hipMemcpy(inputArray, array, sizeof(int) * size, hipMemcpyHostToDevice);
    // bind texture
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
    hipError_t errt = hipBindTexture(0, texture_inputArray, inputArray, channelDesc);
    if (errt != hipSuccess) printf("can not bind inputArray to texture \n");
    errt = hipBindTexture(0, texture_radixArray, radixArray, channelDesc);
    if (errt != hipSuccess) printf("can not bind radixArray to texture \n");

    int max_digit;
    hipMalloc((void **)&largestNum, sizeof(int));
    hipMalloc((void **)&smallestNum, sizeof(int));

    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    reduceMaxMin<<<blockCount, threadCount>>>(g_maxdata, g_mindata);
    reduceMaxMin_Service<<<1, THREADSIZE>>>(g_maxdata, g_mindata, largestNum, smallestNum);

    hipMemcpy(&max, largestNum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, smallestNum, sizeof(int), hipMemcpyDeviceToHost);

    max_digit = max - min;

    while (max_digit / significantDigit > 0) {
        int bucket[RADIX] = {0};

        hipMemcpy(bucketArray, bucket, sizeof(int) * RADIX, hipMemcpyHostToDevice);
        // calcolo frequenza per ogni cifra, questo nel mio blocco.
        histogramKernel<<<blockCount, threadCount>>>(blockBucketArray, radixArray, size, significantDigit, min);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // calcolo la frequenza per ogni cifra, sommando quelle di tutti i block.
        // fondamentalmente sommo all'array delle frequenze il precedente, come facevamo nel vecchio algortimo. A[i-1] = A[i]
        combineBucket<<<1, RADIX>>>(blockBucketArray, bucketArray);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // salva gli indici in cui memorizzare gli elementi ordinati --> fa la magia :D
        indexArrayKernel<<<blockCount, threadCount>>>(bucketArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // salva gli elementi nella corretta posizione ordinati.
        semiSortKernel<<<blockCount, threadCount>>>(indexArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // aggiorno inputArray con il semisortedarray
        copyKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, size);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        significantDigit *= RADIX;
    }
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float transferTime;
    hipEventElapsedTime(&transferTime, start, stop);
    printf("CUDA Time = %.5f ms dim=%d\n", transferTime, size);
    make_csv(0, transferTime, size);
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);

    hipUnbindTexture(texture_inputArray);
}

int main() {
    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int *array = (int *)malloc(size * sizeof(int));
    int i;
    int max_digit = 9999;

    srand(time(NULL));

    for (i = 0; i < size; i++) {
        if (i % 2)
            array[i] = -(rand() % max_digit);
        else
            array[i] = (rand() % max_digit);
    }

    // printf("\nUnsorted List: ");
    // printArray(array, size);

    radixSort(array, size);
    for (int i = 1; i < size; i++)
        if (array[i - 1] > array[i])
            printf("SE SCASSATT O PUNTATOR");

    // printf("\nSorted List:");
    // printArray(array, size);

    printf("\n");

    return 0;
}
