#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define SIZE 8192
#define THREADSIZE 64
#define BLOCKSIZE ((SIZE - 1) / THREADSIZE + 1)
#define RADIX 10
#define FILE_TO_OPEN "OURLASTCODE_shared_measures.csv"

__global__ void copyKernel(int *inArray, int *semiSortArray, int arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < arrayLength) {
        inArray[index] = semiSortArray[index];
    }
}
__global__ void reduceMaxMin(int *g_idata, int *g_maxdata, int *g_mindata) {
    __shared__ int smaxdata[(SIZE / BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(SIZE / BLOCKSIZE)];  // each thread loads one element from global to shared mem unsigned
    int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    smaxdata[tid] = g_idata[i];
    smindata[tid] = g_idata[i];
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem

    if (tid == 0) {
        g_maxdata[blockIdx.x] = smaxdata[0];
        g_mindata[blockIdx.x] = smindata[0];
    }
}

__global__ void reduceMaxMin_Service(int *g_maxdata, int *g_mindata, int *max, int *min) {
    __shared__ int smaxdata[(THREADSIZE)];  // each thread loads one element from global to shared mem unsigned
    __shared__ int smindata[(THREADSIZE)];
    int tid = threadIdx.x;
    if (g_maxdata[tid] > g_maxdata[THREADSIZE + tid])
        smaxdata[tid] = g_maxdata[tid];
    else
        smaxdata[tid] = g_maxdata[THREADSIZE + tid];
    if (g_mindata[tid] < g_mindata[THREADSIZE + tid])
        smindata[tid] = g_mindata[tid];
    else
        smindata[tid] = g_mindata[THREADSIZE + tid];
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem
    if (tid == 0) {
        *max = smaxdata[0];
        *min = smindata[0];
    }
}

__global__ void histogramKernel(int *inArray, int *outArray, int *radixArray, int arrayLength, int significantDigit, int minElement) {
    __shared__ int inArrayShared[THREADSIZE];
    __shared__ int outArrayShared[RADIX];
    __shared__ int radixArrayShared[THREADSIZE];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread = threadIdx.x;
    int blockIndex = blockIdx.x * RADIX;

    int radix;
    int arrayElement;
    int i;

    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArrayShared[i] = 0;
        }
    }

    if (index < arrayLength) {
        inArrayShared[thread] = inArray[index];
    }

    __syncthreads();

    if (index < arrayLength) {
        arrayElement = inArrayShared[thread] - minElement;
        radix = ((arrayElement / significantDigit) % 10);
        radixArrayShared[thread] = radix;
        atomicAdd(&outArrayShared[radix], 1);
    }

    if (index < arrayLength) {
        radixArray[index] = radixArrayShared[thread];
    }
    __syncthreads();
    // forse possimao fare il casino che diventa supermegaultravelocissimo !!!!!!
    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[blockIndex + i] = outArrayShared[i];
        }
    }
}

__global__ void combineBucket(int *blockBucketArray, int *bucketArray) {
    __shared__ int bucketArrayShared[RADIX];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index] = 0;

    for (i = index; i < RADIX * BLOCKSIZE; i = i + RADIX) {
        atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 1; i < RADIX; i++)
            bucketArrayShared[i] += bucketArrayShared[i - 1];
    }
    __syncthreads();
    bucketArray[index] = bucketArrayShared[index];
}

__global__ void indexArrayKernel(int *radixArray, int *bucketArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    int radix;
    int pocket;

    if (index < RADIX) {
        for (i = 0; i < arrayLength; i++) {
            radix = radixArray[arrayLength - i - 1];
            if (radix == index) {
                pocket = --bucketArray[radix];
                indexArray[arrayLength - i - 1] = pocket;
            }
        }
    }
}

__global__ void semiSortKernel(int *inArray, int *outArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;

    if (index < arrayLength) {
        arrayElement = inArray[index];
        arrayIndex = indexArray[index];
        outArray[arrayIndex] = arrayElement;
    }
}

void printArray(int *array, int size) {
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}

int findLargestNum(int *array, int size) {
    int i;
    int largestNum = -1;
    for (i = 0; i < size; i++) {
        if (array[i] > largestNum)
            largestNum = array[i];
    }
    return largestNum;
}
void make_csv(float gflops, float time, float N) {
    FILE *fp;
    if (access(FILE_TO_OPEN, F_OK) == 0) {
        fp = fopen(FILE_TO_OPEN, "a");

    } else {
        fp = fopen(FILE_TO_OPEN, "w");
        fprintf(fp, "N, gflops, time_sec\n");
    }
    fprintf(fp, "%f, %f, %.5f\n", N, gflops, time);
    fclose(fp);
}

void radixSort(int *array, int size) {
    int significantDigit = 1;
    hipEvent_t start, stop;
    int threadCount;
    int blockCount;

    int min, max;

    threadCount = THREADSIZE;
    blockCount = BLOCKSIZE;

    int *outputArray;
    int *inputArray;
    int *radixArray;
    int *bucketArray;
    int *indexArray;
    int *semiSortArray;
    int *blockBucketArray;
    int *g_maxdata;
    int *g_mindata;
    int *largestNum;
    int *smallestNum;

    hipMalloc((void **)&inputArray, sizeof(int) * size);
    hipMalloc((void **)&indexArray, sizeof(int) * size);

    hipMalloc((void **)&g_maxdata, sizeof(int) * BLOCKSIZE);
    hipMalloc((void **)&g_mindata, sizeof(int) * BLOCKSIZE);

    hipMalloc((void **)&radixArray, sizeof(int) * size);

    hipMalloc((void **)&outputArray, sizeof(int) * size);

    hipMalloc((void **)&semiSortArray, sizeof(int) * size);
    hipMalloc((void **)&bucketArray, sizeof(int) * RADIX);
    hipMalloc((void **)&blockBucketArray, sizeof(int) * RADIX * BLOCKSIZE);

    hipMemcpy(inputArray, array, sizeof(int) * size, hipMemcpyHostToDevice);

    int max_digit;
    hipMalloc((void **)&largestNum, sizeof(int));
    hipMalloc((void **)&smallestNum, sizeof(int));

    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    reduceMaxMin<<<blockCount, threadCount>>>(inputArray, g_maxdata, g_mindata);
    reduceMaxMin_Service<<<1, THREADSIZE>>>(g_maxdata, g_mindata, largestNum, smallestNum);

    hipMemcpy(&max, largestNum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, smallestNum, sizeof(int), hipMemcpyDeviceToHost);

    max_digit = max - min;

    while (max_digit / significantDigit > 0) {
        int bucket[RADIX] = {0};
        hipMemcpy(bucketArray, bucket, sizeof(int) * RADIX, hipMemcpyHostToDevice);
        // calcolo frequenza per ogni cifra, questo nel mio blocco.
        histogramKernel<<<blockCount, threadCount>>>(inputArray, blockBucketArray, radixArray, size, significantDigit, min);
        hipDeviceSynchronize();
        // calcolo la frequenza per ogni cifra, sommando quelle di tutti i block.
        // fondamentalmente sommo all'array delle frequenze il precedente, come facevamo nel vecchio algortimo. A[i-1] = A[i]
        combineBucket<<<1, RADIX>>>(blockBucketArray, bucketArray);
        hipDeviceSynchronize();
        // salva gli indici in cui memorizzare gli elementi ordinati --> fa la magia :D
        indexArrayKernel<<<blockCount, threadCount>>>(radixArray, bucketArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        // salva gli elementi nella corretta posizione ordinati.
        semiSortKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        // aggiorno inputArray con il semisortedarray
        copyKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, size);
        hipDeviceSynchronize();

        significantDigit *= RADIX;
    }
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float transferTime;
    hipEventElapsedTime(&transferTime, start, stop);
    printf("CUDA Time = %.5f ms dim=%d\n", transferTime, size);
    make_csv(0, transferTime, size);
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);
}

int main() {
    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int array[size];
    int i;
    int max_digit = 9999;

    srand(time(NULL));

    for (i = 0; i < size; i++) {
        if (i % 2)
            array[i] = -(rand() % max_digit);
        else
            array[i] = (rand() % max_digit);
    }

    // printf("\nUnsorted List: ");
    // printArray(array, size);

    radixSort(array, size);
    for (int i = 1; i < size; i++)
        if (array[i - 1] > array[i])
            printf("SE SCASSATT O PUNTATOR");

    // printf("\nSorted List:");
    // printArray(array, size);

    printf("\n");

    return 0;
}