#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define CUDA_CHECK(X)                                                     \
    {                                                                     \
        hipError_t _m_cudaStat = X;                                      \
        if (hipSuccess != _m_cudaStat) {                                 \
            fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
                    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#ifndef SIZE
#define SIZE 8192 * 12
#endif

#ifndef THREADSIZE
#define THREADSIZE 1024
#endif

#ifndef MAX_DIGIT
#define MAX_DIGIT 9999
#endif

#define BLOCKSIZE ((SIZE - 1) / THREADSIZE + 1)
#define RADIX 10
#define FILE_TO_OPEN "Global_measures.csv"

__global__ void copyKernel(int *inArray, int *semiSortArray, int arrayLength) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < arrayLength) {
        inArray[index] = semiSortArray[index];
    }
}
__global__ void reduceMaxMin(int *g_idata, int *g_maxdata, int *g_mindata, int *smaxdata, int *smindata) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    smaxdata[i] = g_idata[i];
    smindata[i] = g_idata[i];
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            if (smaxdata[i + s] > smaxdata[i]) {
                smaxdata[i] = smaxdata[i + s];
            }
            if (smindata[i + s] < smindata[i]) {
                smindata[i] = smindata[i + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem

    if (threadIdx.x == 0) {
        g_maxdata[blockIdx.x] = smaxdata[blockIdx.x * blockDim.x];
        g_mindata[blockIdx.x] = smindata[blockIdx.x * blockDim.x];
    }
}

__global__ void reduceMaxMin_Service(int *g_maxdata, int *g_mindata, int *max, int *min, int *smaxdata, int *smindata) {
    int tid = threadIdx.x;
    smaxdata[tid] = g_maxdata[tid];
    smindata[tid] = g_mindata[tid];
    for (unsigned int s = 1; s < BLOCKSIZE / THREADSIZE; s++) {
        int index = THREADSIZE * s + tid;
        if (smaxdata[tid] < g_maxdata[index])
            smaxdata[tid] = g_maxdata[index];
        if (smindata[tid] > g_mindata[index])
            smindata[tid] = g_mindata[index];
    }
    __syncthreads();  // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smaxdata[tid + s] > smaxdata[tid]) {
                smaxdata[tid] = smaxdata[tid + s];
            }
            if (smindata[tid + s] < smindata[tid]) {
                smindata[tid] = smindata[tid + s];
            }
        }
        __syncthreads();
    }  // write result for this block to global mem
    if (tid == 0) {
        *max = smaxdata[0];
        *min = smindata[0];
    }
}

__global__ void histogramKernel(int *inArray, int *outArray, int *radixArray, int arrayLength, int significantDigit, int minElement, int *inArrayShared, int *outArrayShared, int *radixArrayShared) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread = threadIdx.x;
    int blockIndex = blockIdx.x * RADIX;

    int radix;
    int arrayElement;
    int i;

    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[i] = 0;
        }
    }

    if (index < arrayLength) {
        inArrayShared[thread] = inArray[index];
    }

    __syncthreads();

    if (index < arrayLength) {
        arrayElement = inArrayShared[thread] - minElement;
        radix = ((arrayElement / significantDigit) % 10);
        radixArrayShared[thread] = radix;
        atomicAdd(&outArray[blockIndex + radix], 1);
    }

    if (index < arrayLength) {
        radixArray[index] = radixArrayShared[thread];
    }
    __syncthreads();
    // forse possimao fare il casino che diventa supermegaultravelocissimo !!!!!!
    if (thread == 0) {
        for (i = 0; i < RADIX; i++) {
            outArray[blockIndex + i] = outArrayShared[i];
        }
    }
}

__global__ void combineBucket(int *blockBucketArray, int *bucketArray, int *bucketArrayShared) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;

    bucketArrayShared[index] = 0;

    for (i = index; i < RADIX * BLOCKSIZE; i = i + RADIX) {
        atomicAdd(&bucketArrayShared[index], blockBucketArray[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (i = 1; i < RADIX; i++)
            bucketArrayShared[i] += bucketArrayShared[i - 1];
    }
    __syncthreads();
    bucketArray[index] = bucketArrayShared[index];
}

__global__ void indexArrayKernel(int *radixArray, int *bucketArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    int radix;
    int pocket;

    if (index < RADIX) {
        for (i = 0; i < arrayLength; i++) {
            radix = radixArray[arrayLength - i - 1];
            if (radix == index) {
                pocket = --bucketArray[radix];
                indexArray[arrayLength - i - 1] = pocket;
            }
        }
    }
}

__global__ void semiSortKernel(int *inArray, int *outArray, int *indexArray, int arrayLength, int significantDigit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int arrayElement;
    int arrayIndex;

    if (index < arrayLength) {
        arrayElement = inArray[index];
        arrayIndex = indexArray[index];
        outArray[arrayIndex] = arrayElement;
    }
}

void printArray(int *array, int size) {
    int i;
    printf("[ ");
    for (i = 0; i < size; i++)
        printf("%d ", array[i]);
    printf("]\n");
}

int findLargestNum(int *array, int size) {
    int i;
    int largestNum = -1;
    for (i = 0; i < size; i++) {
        if (array[i] > largestNum)
            largestNum = array[i];
    }
    return largestNum;
}

void make_csv(float gflops, float time, float N) {
    FILE *fp;
    if (access(FILE_TO_OPEN, F_OK) == 0) {
        fp = fopen(FILE_TO_OPEN, "a");

    } else {
        fp = fopen(FILE_TO_OPEN, "w");
        fprintf(fp, "N, BlockSize, GridSize, gflops, time_sec\n");
    }
    fprintf(fp, "%f, %d, %d, %f, %.5f\n", N, THREADSIZE, BLOCKSIZE, gflops, time / 1000);
    fclose(fp);
}

void radixSort(int *array, int size) {
    int significantDigit = 1;
    hipEvent_t start, stop;
    int threadCount;
    int blockCount;

    int min, max;

    threadCount = THREADSIZE;
    blockCount = BLOCKSIZE;

    int *outputArray;
    int *inputArray;
    int *radixArray;
    int *bucketArray;
    int *indexArray;
    int *semiSortArray;
    int *blockBucketArray;
    int *g_maxdata;
    int *g_mindata;
    int *largestNum;
    int *smallestNum;

    int *inArrayShared;
    int *outArrayShared;
    int *radixArrayShared;
    int *smaxdata;
    int *smindata;

    int *bucketArrayShared;

    CUDA_CHECK(hipMalloc((void **)&bucketArrayShared, sizeof(int) * RADIX));

    CUDA_CHECK(hipMalloc((void **)&inArrayShared, sizeof(int) * THREADSIZE));
    CUDA_CHECK(hipMalloc((void **)&outArrayShared, sizeof(int) * RADIX));
    CUDA_CHECK(hipMalloc((void **)&radixArrayShared, sizeof(int) * THREADSIZE));

    CUDA_CHECK(hipMalloc((void **)&inputArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&indexArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&smaxdata, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&smindata, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&g_maxdata, sizeof(int) * BLOCKSIZE));
    CUDA_CHECK(hipMalloc((void **)&g_mindata, sizeof(int) * BLOCKSIZE));

    CUDA_CHECK(hipMalloc((void **)&radixArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&outputArray, sizeof(int) * size));

    CUDA_CHECK(hipMalloc((void **)&semiSortArray, sizeof(int) * size));
    CUDA_CHECK(hipMalloc((void **)&bucketArray, sizeof(int) * RADIX));
    CUDA_CHECK(hipMalloc((void **)&blockBucketArray, sizeof(int) * RADIX * BLOCKSIZE));

    hipMemcpy(inputArray, array, sizeof(int) * size, hipMemcpyHostToDevice);

    int max_digit;
    hipMalloc((void **)&largestNum, sizeof(int));
    hipMalloc((void **)&smallestNum, sizeof(int));

    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    reduceMaxMin<<<blockCount, threadCount>>>(inputArray, g_maxdata, g_mindata, smaxdata, smindata);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    reduceMaxMin_Service<<<1, THREADSIZE>>>(g_maxdata, g_mindata, largestNum, smallestNum, smaxdata, smindata);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }

    hipMemcpy(&max, largestNum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min, smallestNum, sizeof(int), hipMemcpyDeviceToHost);

    max_digit = max - min;
    while (max_digit / significantDigit > 0) {
        int bucket[RADIX] = {0};
        hipMemcpy(bucketArray, bucket, sizeof(int) * RADIX, hipMemcpyHostToDevice);
        // calcolo frequenza per ogni cifra, questo nel mio blocco.
        histogramKernel<<<blockCount, threadCount>>>(inputArray, blockBucketArray, radixArray, size, significantDigit, min, inArrayShared, outArrayShared, radixArrayShared);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // calcolo la frequenza per ogni cifra, sommando quelle di tutti i block.
        // fondamentalmente sommo all'array delle frequenze il precedente, come facevamo nel vecchio algortimo. A[i-1] = A[i]
        combineBucket<<<1, RADIX>>>(blockBucketArray, bucketArray, bucketArrayShared);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // salva gli indici in cui memorizzare gli elementi ordinati --> fa la magia :D
        indexArrayKernel<<<blockCount, threadCount>>>(radixArray, bucketArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // salva gli elementi nella corretta posizione ordinati.
        semiSortKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, indexArray, size, significantDigit);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }
        // aggiorno inputArray con il semisortedarray
        copyKernel<<<blockCount, threadCount>>>(inputArray, semiSortArray, size);
        hipDeviceSynchronize();
        mycudaerror = hipGetLastError();
        if (mycudaerror != hipSuccess) {
            fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
            exit(1);
        }

        significantDigit *= RADIX;
    }
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float transferTime;
    hipEventElapsedTime(&transferTime, start, stop);
    printf("CUDA Time = %.5f ms dim=%d\n", transferTime, size);

    make_csv(0, transferTime, size);
    hipMemcpy(array, inputArray, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(inputArray);
    hipFree(indexArray);
    hipFree(radixArray);
    hipFree(bucketArray);
    hipFree(blockBucketArray);
    hipFree(outputArray);
    hipFree(semiSortArray);
}

int main() {
    printf("\n\nRunning Radix Sort Example in C!\n");
    printf("----------------------------------\n");

    int size = SIZE;
    int *array = (int *)malloc(size * sizeof(int));
    int i;
    srand(time(NULL));

    for (i = 0; i < size; i++) {
        if (i % 2)
            array[i] = -(rand() % MAX_DIGIT);
        else
            array[i] = (rand() % MAX_DIGIT);
    }

    // printf("\nUnsorted List: ");
    // printArray(array, size);

    radixSort(array, size);
    for (int i = 1; i < size; i++)
        if (array[i - 1] > array[i])
            printf("SE SCASSATT O PUNTATOR");

    // printf("\nSorted List:");
    // printArray(array, size);

    printf("\n");

    return 0;
}